#include "hip/hip_runtime.h"

#define CudaGlobal __global__
#define CudaDevice __device__
#define CudaHost __host__

__device__ float box_iou()
{
}

__global__ void nms_kernel(float *bboxes, int num_bboxes, float thr)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int count = bboxes[0];
    if (tid >= count)
        return;
}

__global__ void yolo_decode_kernel(float* predict,int num_bboxes,float conf,float* parray,int max_objects)
{
    int tid=blockDim.x*blockIdx.x+threadIdx.x;
    int count=predict[0];
    if(tid>=count)
        return;

    float* pitem=predict+1+tid*(/sizeof(float));
}